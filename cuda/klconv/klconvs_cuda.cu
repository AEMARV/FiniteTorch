#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <math.h>
#include </usr/local/cuda-9.0/targets/x86_64-linux/include/hiprand/hiprand_kernel.h>
// Kernels      ***********************
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



__device__ __forceinline__ size_t calcLinInd(int h, int w, int c, int b, size_t HD, size_t WD,size_t CD ){

    return w + WD*(h + HD*(c + CD*(b)));
}
__device__ __forceinline__ void calcIndpInd(int linind, int h, int w, int c, int b, size_t HD, size_t WD,size_t CD,int *out ){
*out = w + WD*(h + HD*(c + CD*(b)));
    return ;
}
__device__ __forceinline__ void calcIndForRandom(int linind, int h, int w, int c, int b, size_t HD, size_t WD,size_t CD,int *out ){
*out = w + WD*(h + HD*(c + CD*(b)));
    return ;
}
__device__ __forceinline__ size_t calcLinInd4(int idx_a, int idx_b, int idx_c, int idx_d, size_t AD, size_t BD,size_t CD, size_t DD ){

    return idx_d + DD*(idx_c + CD*(idx_b + BD*(idx_a)));
}

/*
template <typename scalar_t>
__global__ void klconvs_cuda_forward_kernel_new(
    unsigned int seed,
    const scalar_t *input,
    const scalar_t *p_filt,
    scalar_t *out,
    const scalar_t *pad,
    const size_t filt_h,
    const size_t filt_w,
    const size_t filt_c,
    const size_t filt_n,
    const size_t inp_h,
    const size_t inp_w,
    const size_t inp_c,
    const size_t inp_b
    ){
const int out_w_sz = blockDim.x, out_h_sz = blockDim.y;
const int out_chan_sz = gridDim.x, batchsz = gridDim.y;
const int w = threadIdx.x , h = threadIdx.y;
const int out_chan_id = blockIdx.x, batch_id=  blockIdx.y;
const int out_idx = threadIdx.x + blockDim.x*(threadIdx.y + blockDim.y*(blockIdx.x + gridDim.x*(blockIdx.y)));
hiprandState_t state;
//for( int channel =0 : channel < )
float temp_reg;
float this_px_out = 0;
bool flag = true;
bool isoutboundh = false;
bool isoutbound = false;
float randnum = 0;
int input_idx = 0;
int filt_idx = 0;
for (int dh=0 ; dh< out_h_sz; dh++){
    isoutboundh = dh + h > inp_h;
    for ( int dw = 0 ; dw< out_w_sz; dw++ ){
        isoutbound = isoutboundh | dw + w > inp_w;
        hiprand_init(seed,out_idx,dw + dh*(out_w_sz),&state);
        randnum = hiprand_uniform(&state);
        flag = true;
        for  ( int chan = 0 ; chan < out_chan_sz; chan++){
            // find the correct index of filt
            // get the index val from input
            // add to final answer;
            calcLinInd(dh,dw,chan,out_chan_id,filt_h,filt_w,filt_c, &filt_idx);//[out_chan_id][chan][dh][dw]
            temp_reg = p_filt[filt_idx];
            if (temp_reg > randnum && flag) {
                if (!isoutbound){
                calcLinInd(h+dh ,w+dw ,chan,batch_id, inp_h, inp_w, inp_c, &input_idx);
                this_px_out += input[input_idx];
                }
                flag = false;
            }
            else{
                randnum = randnum - temp_reg;
            }

        }


    }


}
out[out_idx] = this_px_out;


hiprand_init(seed,0,1,&state);
float randnum = hiprand_uniform(&state);
hiprand_init(seed+1,0,0,&state);
float randnum2 = hiprand_uniform(&state);
printf("time : %f , %f --\n",randnum, randnum2);

//printf("out_idx %d: left to right %d,%d,%d,%d   with dims %d,%d,%d,%d\n", out_idx, threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
///__shared__ float[32][32] input_tile; // define shared memory
}
*/
template <typename scalar_t>
__global__ void klconvs_cuda_forward_kernel(
    const scalar_t* __restrict__ input,
    const float* __restrict__ l_filt,
    scalar_t* __restrict__ out,
    scalar_t* __restrict__ random,
    const size_t  filt_h,
    const size_t filt_w,
    const size_t filt_n,
    const size_t inp_h,
    const size_t inp_w,
    const size_t inp_c,
    const size_t inp_b,
    const int totalOutPx,
    const int totalTreads
    ){
    const size_t threadlinidx = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadlinidx < totalOutPx){

        // Calculate Imout Indices
        int int_temp_reg;
        const int im_w_idx = threadlinidx % inp_w;
        int_temp_reg =inp_w;
        const int im_h_idx = (threadlinidx/int_temp_reg) % inp_h;
        int_temp_reg *=inp_h;
        const int im_c_idx = (threadlinidx/(int_temp_reg)) % filt_n;
        int_temp_reg *=filt_n;
        const int im_b_idx = (threadlinidx/(int_temp_reg));
        size_t size_temp_reg;
        float randnum;
        float this_px_out=0;
        float float_temp;
        float log_probs=0;
        size_t input_idx=0;
        int dh;
        int dw;
        int chan;
        int cur_im_h =0;
        int cur_im_w =0;
        int rand_idx = 0;
        float p_filt;
        for ( dh= 0 ; dh < filt_h; dh++){
            cur_im_h= dh + im_h_idx - ((filt_h)/2);
            if (cur_im_h< 0){
                    continue;
                            }
            if (cur_im_h >= inp_h){
                    break;
                }

            for (dw = 0 ; dw < filt_w; dw++ ){
                cur_im_w = dw + im_w_idx - ((filt_w)/2);
                if (cur_im_w<0){
                    continue;
                }
                if (cur_im_w >= inp_w){
                    break;
                }
                randnum = random[threadlinidx + totalOutPx*(dw +  filt_w*( dh ))];// * isoutbound;  // GLOBAL MEM ACCESS
                for  ( chan = 0 ; chan < inp_c; chan++){
                    // find the correct index of filt
                    // get the index val from input
                    // add to final answer;
                    size_temp_reg = calcLinInd( dh, dw, chan,im_c_idx, filt_h, filt_w, inp_c);
                    float_temp = l_filt[size_temp_reg];
                    p_filt = expf(float_temp);
                    if (randnum <= p_filt){
                        input_idx = calcLinInd(cur_im_h, cur_im_w, chan, im_b_idx, inp_h, inp_w, inp_c);
                        //inp_indices[dh][dw] = input_idx; ////////// GL MEM ACCESS  ////*********** check wether bool*float is float
                        //randnum = 100;
                        log_probs = float_temp;
                        random[threadlinidx + totalOutPx*(dw +  filt_w*( dh ))] = __int2float_rn(chan);
                        break;
                    }
                    //flag = (flag && !flag2);
                    //j = j + flag2;
                    randnum = randnum - p_filt;
                }
                this_px_out += input[input_idx] -log_probs;
            }
        }

        //for (int j=0 ; j< input_idx; j++){
        //    this_px_out += input[inp_indices[j]];
        //}
        out[threadlinidx] = this_px_out; //////////////////// GL MEM WRITE

    }
}

/*
template <typename scalar_t>
__global__ void klconvs_cuda_forward_kernel_single_loop(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ p_filt,
    scalar_t* __restrict__ out,
    const scalar_t* __restrict__ random,
    const size_t  filt_h,
    const size_t filt_w,
    const size_t filt_c,
    const size_t filt_n,
    const size_t inp_h,
    const size_t inp_w,
    const size_t inp_c,
    const size_t inp_b,
    const int totalOutPx,
    const int totalThreads
    ){
    const int threadlinidx = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadlinidx < totalThreads){
        // Calculate Imout Indices
        int int_temp_reg =1;
        const int im_w_idx = threadlinidx % inp_w;
        int_temp_reg = inp_w;
        const int im_h_idx = (threadlinidx/int_temp_reg) % inp_h;
        int_temp_reg = int_temp_reg * inp_h;
        const int im_c_idx = (threadlinidx/(int_temp_reg)) % filt_n;
        int_temp_reg = int_temp_reg * filt_n;
        const int im_b_idx = (threadlinidx/(int_temp_reg)) %inp_b;
        int_temp_reg = int_temp_reg * inp_b;
        const int dw = (threadlinidx/(int_temp_reg)) %filt_w;
        int_temp_reg = int_temp_reg * filt_w;
        const int dh = (threadlinidx/(int_temp_reg)) %filt_h;
        int out_idx = im_w_idx;
        int_temp_reg= inp_w;
        out_idx += im_h_idx * int_temp_reg;
        int_temp_reg *= inp_h;
        out_idx += im_c_idx * int_temp_reg;
        int_temp_reg *= inp_c;
        out_idx += im_b_idx * int_temp_reg;
        float randnum ;
        int rand_idx;
        float temp_reg=0;
        float this_px_out=0;
        // Flags
        bool flag = true;
        bool flag2= false;
        bool isoutboundh = false;
        bool isoutbound = false;

        int input_idx;
        int filt_idx;
        int j = 0;


        isoutboundh = (dh + im_h_idx) > inp_h;
        isoutbound = isoutboundh || ((dw + im_w_idx) > inp_w);
        rand_idx = threadlinidx + totalOutPx*(dw+ filt_w*(dh));
        randnum = random[rand_idx];// * isoutbound;  // GLOBAL MEM ACCESS
        flag = true;
        for  ( int chan = 0 ; chan < filt_c; chan++){
            // find the correct index of filt
            // get the index val from input
            // add to final answer;
            calcLinInd(dh,dw,chan,im_c_idx,filt_h,filt_w,filt_c, &filt_idx);//[out_chan_id][chan][dh][dw]
            //temp_reg = p_filt[filt_idx];  ////////////GLOBAL MEM ACESSS
            flag2 = flag && (temp_reg >= randnum) && (!isoutbound);
            calcLinInd(im_h_idx + dh, im_w_idx + dw, chan, im_b_idx, inp_h, inp_w, inp_c, &input_idx);
            this_px_out = this_px_out + (flag2 * input[input_idx]); ////////// GL MEM ACESSS  ////*********** check wether bool*float is float
            if (flag2){
                break;
            }
            flag = (flag && !flag2);
            j = j + flag2;
            randnum = randnum - temp_reg;
        }

        //atomicAdd(&(out[out_idx]),this_px_out);
       // out[out_idx] += this_px_out; //////////////////// GL MEM WRITE

    }
}

template <typename scalar_t>
__global__ void klconvs_cuda_forward_kernel_zero_loop(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ p_filt,
    scalar_t* out,
    const scalar_t* __restrict__ random,
    const size_t  filt_h,
    const size_t filt_w,
    const size_t filt_c,
    const size_t filt_n,
    const size_t inp_h,
    const size_t inp_w,
    const size_t inp_c,
    const size_t inp_b,
    const long int totalOutPx,
    const long int totalActiveThreads
    ){
    const long int threadlinidx = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadlinidx < totalActiveThreads){
        // Calculate Imout Indices
        int int_temp_reg =1;
        const int im_w_idx = threadlinidx % inp_w;
        int_temp_reg = inp_w;
        const int im_h_idx = (threadlinidx/int_temp_reg) % inp_h;
        int_temp_reg = int_temp_reg * inp_h;
        const int im_c_idx = (threadlinidx/(int_temp_reg)) % filt_n;
        int_temp_reg = int_temp_reg * filt_n;
        const int im_b_idx = (threadlinidx/(int_temp_reg)) % inp_b;
        int_temp_reg = int_temp_reg * inp_b;
        const int dw = (threadlinidx/(int_temp_reg)) % filt_w;
        int_temp_reg = int_temp_reg * filt_w;
        const int dh = (threadlinidx/(int_temp_reg)) % filt_h;
        int_temp_reg *= filt_h;
        int chan = (threadlinidx/(int_temp_reg));
        int out_idx = im_w_idx + inp_w*( im_h_idx + inp_h*(im_c_idx + filt_n*(im_b_idx + inp_b*(dw + filt_w*(dh )))));

        float randnum=0.5 ;
        long int rand_idx;
        float current_cumprob_reg=0;
        float prev_cumprob_reg=0;
        // Flags
        bool flag = true;
        bool flag2= false;
        bool isoutboundh = false;
        bool isoutbound = false;

        long int input_idx;
        long int filt_idx;
        long int prev_filt_idx;
        int j = 0;


        isoutboundh = (dh + im_h_idx) >= inp_h;
        isoutbound = isoutboundh || ((dw + im_w_idx) >= inp_w);
        rand_idx = threadlinidx % (inp_w*inp_h*filt_n*inp_b*filt_w*filt_h);
        randnum = random[rand_idx];// * isoutbound;  // GLOBAL MEM ACCESS
        flag = true;

        // find the correct index of filt
        // get the index val from input
        // add to final answer;

        calcLinInd(dh,dw,chan,im_c_idx,filt_h,filt_w,filt_c, &filt_idx);//[out_chan_id][chan][dh][dw]

        current_cumprob_reg = p_filt[filt_idx];  ////////////GLOBAL MEM ACESSS
        if (chan == 0){
            prev_cumprob_reg = 0 ;
        }
        else{
            calcLinInd(dh,dw,(chan-1),im_c_idx,filt_h,filt_w,filt_c, &prev_filt_idx);//[out_chan_id][chan][dh][dw]
            prev_cumprob_reg = p_filt[prev_filt_idx];  ////////////GLOBAL MEM ACESSS
        }
        flag2 = (prev_cumprob_reg < randnum) && (current_cumprob_reg >= randnum) && (!isoutbound);

        //this_px_out =  input[input_idx];
        if (flag2){
            //out_idx = 0;
            calcLinInd(im_h_idx + dh, im_w_idx + dw, chan, im_b_idx, inp_h, inp_w, inp_c, &input_idx);

            //out[out_idx] = atomicAdd(&(out[out_idx]),input[input_idx];////////// GL MEM ACESSS  ////*********** check wether bool*float is float
            if (out_idx > filt_h *filt_w *totalOutPx){
                printf("Culprit: %d", out_idx);
            }
              out[out_idx] += input[input_idx]; //////////////////// GL MEM WRITE

        }

        //atomicAdd(&(out[out_idx]),this_px_out);
       // out[out_idx] += this_px_out; //////////////////// GL MEM WRITE

    }
}
/* ----------------------   Print Tests
printf("number of times went in %d\n",j);
hiprand_init(seed,0,1,&state);
float randnum = hiprand_uniform(&state);
hiprand_init(seed+1,0,0,&state);
float randnum2 = hiprand_uniform(&state);
printf("time : %f , %f --\n",randnum, randnum2);
printf("out_idx %d: left to right %d,%d,%d,%d   with dims %d,%d,%d,%d\n", out_idx, threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
*/

//__shared__ float[32][32] input_tile; // define shared memory


template <typename scalar_t>
__global__ void klconvs_cuda_backward_kernel(
    const scalar_t* __restrict__ input, //TODO: MAKE sure the dims are dzdin and the threads are compatible
    const scalar_t* __restrict__ lfilt,
    const scalar_t* __restrict__ dzdout,
    const scalar_t* __restrict__ random,
    float* __restrict__ dzdin,
    float* __restrict__ dzdl_filt,
    const size_t  filt_h,
    const size_t filt_w,
    const size_t filt_n,
    const size_t inp_h,
    const size_t inp_w,
    const size_t inp_c,
    const size_t inp_b,
    const int totalThreads
){
    const size_t threadlinidx = blockIdx.x*blockDim.x + threadIdx.x;
    int int_temp_reg;
    const int im_w_idx = threadlinidx % inp_w;
    int_temp_reg =inp_w;
    const int im_h_idx = (threadlinidx/int_temp_reg) % inp_h;
    int_temp_reg *=inp_h;
    const int filt_n_idx = (threadlinidx/(int_temp_reg)) % filt_n;
    int_temp_reg *=filt_n;
    const int im_b_idx = (threadlinidx/(int_temp_reg)) % inp_b;
    int_temp_reg *= inp_b;
    const int filt_w_idx = (threadlinidx/(int_temp_reg)) % filt_w;
    int_temp_reg *= filt_w;
    const int filt_h_idx = (threadlinidx/(int_temp_reg)) ;

    const int inp_w_idx = im_w_idx + filt_w_idx - (filt_w/2) ;
    const int inp_h_idx = im_h_idx + filt_h_idx - (filt_h/2) ;
    bool flag = threadlinidx < totalThreads && inp_w_idx >=0 && inp_w_idx <inp_w && inp_h_idx >=0 && inp_h_idx <inp_h;
    if (flag){
        // Calculate Imout Indices

        float randnum;
        float this_px_out=0;
        float float_temp;
        // Flags
       // bool flag = true;
       // bool flag2= false;
       // bool isoutboundh = false;
       // bool isoutbound = false;
        int chan  = __float2int_rn(random[threadlinidx]);
        int input_idx=0;
        //int filt_idx;
        //int j = 0;

        size_t dzdout_idx = calcLinInd4(im_b_idx,filt_n_idx,im_h_idx,im_w_idx ,inp_b, filt_n, inp_h, inp_w);
        size_t dzdin_idx = calcLinInd4(im_b_idx , chan , inp_h_idx , inp_w_idx , inp_b, inp_c, inp_h, inp_w);
//       printf("%d \n",chan);
        //size_t dzdin_idx = im_b_idx + chan + inp_h_idx + inp_w_idx;
        size_t dzdl_filt_idx = calcLinInd4(filt_n_idx , chan , filt_h_idx , filt_w_idx, filt_n, inp_c, filt_h, filt_w );
        float dzdoutthis = dzdout[dzdout_idx];
//dzdin[0] += dzdoutthis;
        //dzdin[dzdin_idx] += dzdoutthis;
        atomicAdd(&(dzdin[dzdin_idx]), dzdoutthis);
        atomicAdd(&(dzdl_filt[dzdl_filt_idx]), dzdoutthis* (input[dzdin_idx] - 1 - lfilt[dzdl_filt_idx])); // p differentiable grad
        //atomicAdd(&(dzdl_filt[dzdl_filt_idx]), -dzdoutthis); // p NON-differentiable grad
        //dzdl_filt[dzdl_filt_idx] += dzdoutthis* input[dzdin_idx] ;
    }



}

template <typename scalar_t>
__global__ void klconv_cuda_backward_kernel(
    const scalar_t* __restrict__ input, //TODO: MAKE sure the dims are dzdin and the threads are compatible
    const scalar_t* __restrict__ lfilt,
    const scalar_t* __restrict__ dzdout,
    const scalar_t* __restrict__ random,
    float* __restrict__ dzdin,
    float* __restrict__ dzdl_filt,
    const size_t  filt_h,
    const size_t filt_w,
    const size_t filt_n,
    const size_t inp_h,
    const size_t inp_w,
    const size_t inp_c,
    const size_t inp_b,
    const int totalThreads
){
    const size_t threadlinidx = blockIdx.x*blockDim.x + threadIdx.x;
    int int_temp_reg;
    const int im_w_idx = threadlinidx % inp_w;
    int_temp_reg =inp_w;
    const int im_h_idx = (threadlinidx/int_temp_reg) % inp_h;
    int_temp_reg *=inp_h;
    const int filt_n_idx = (threadlinidx/(int_temp_reg)) % filt_n;
    int_temp_reg *=filt_n;
    const int im_b_idx = (threadlinidx/(int_temp_reg)) % inp_b;
    int_temp_reg *= inp_b;
    const int filt_w_idx = (threadlinidx/(int_temp_reg)) % filt_w;
    int_temp_reg *= filt_w;
    const int filt_h_idx = (threadlinidx/(int_temp_reg)) ;

    const int inp_w_idx = im_w_idx + filt_w_idx - (filt_w/2) ;
    const int inp_h_idx = im_h_idx + filt_h_idx - (filt_h/2) ;
    bool flag = threadlinidx < totalThreads && inp_w_idx >=0 && inp_w_idx <inp_w && inp_h_idx >=0 && inp_h_idx <inp_h;
    if (flag){
        // Calculate Imout Indices

        float randnum;
        float this_px_out=0;
        float float_temp;
        int chan  = __float2int_rn(random[threadlinidx]);
        int input_idx=0;
        float dzdoutthis;
        float cur_lfilt;
        float cur_pfilt;
        float cur_in;

        for (chan =0 ; chan < inp_c ; chan++){
            size_t dzdout_idx = calcLinInd4(im_b_idx,filt_n_idx,im_h_idx,im_w_idx ,inp_b, filt_n, inp_h, inp_w);
            size_t dzdin_idx = calcLinInd4(im_b_idx , chan , inp_h_idx , inp_w_idx , inp_b, inp_c, inp_h, inp_w);
            size_t dzdl_filt_idx = calcLinInd4(filt_n_idx , chan , filt_h_idx , filt_w_idx, filt_n, inp_c, filt_h, filt_w );

            cur_lfilt =  lfilt[dzdl_filt_idx];
            cur_pfilt = expf(cur_lfilt);
            cur_in = input[dzdin_idx];

            dzdoutthis = dzdout[dzdout_idx];
            atomicAdd(&(dzdin[dzdin_idx]), (dzdoutthis*cur_pfilt));
            atomicAdd(&(dzdl_filt[dzdl_filt_idx]), dzdoutthis * cur_pfilt* (cur_in - 1 - cur_lfilt)); // p differentiable grad
        }
    }



}


template <typename scalar_t>
__global__ void klconvs_cuda_backward_rand_kernel(
    const scalar_t* __restrict__ input, //TODO: MAKE sure the dims are dzdin and the threads are compatible
    const scalar_t* __restrict__ lfilt,
    const scalar_t* __restrict__ dzdout,
    const scalar_t* __restrict__ random,
    float* __restrict__ dzdin,
    float* __restrict__ dzdl_filt,
    const size_t  filt_h,
    const size_t filt_w,
    const size_t filt_n,
    const size_t inp_h,
    const size_t inp_w,
    const size_t inp_c,
    const size_t inp_b,
    const int totalThreads
){
    const size_t threadlinidx = blockIdx.x*blockDim.x + threadIdx.x;
    int int_temp_reg;
    const int im_w_idx = threadlinidx % inp_w;
    int_temp_reg =inp_w;
    const int im_h_idx = (threadlinidx/int_temp_reg) % inp_h;
    int_temp_reg *=inp_h;
    const int filt_n_idx = (threadlinidx/(int_temp_reg)) % filt_n;
    int_temp_reg *=filt_n;
    const int im_b_idx = (threadlinidx/(int_temp_reg)) % inp_b;
    int_temp_reg *= inp_b;
    const int filt_w_idx = (threadlinidx/(int_temp_reg)) % filt_w;
    int_temp_reg *= filt_w;
    const int filt_h_idx = (threadlinidx/(int_temp_reg)) ;

    const int inp_w_idx = im_w_idx + filt_w_idx - (filt_w/2) ;
    const int inp_h_idx = im_h_idx + filt_h_idx - (filt_h/2) ;
    bool flag = threadlinidx < totalThreads && inp_w_idx >=0 && inp_w_idx <inp_w && inp_h_idx >=0 && inp_h_idx <inp_h;
    if (flag){
        // Calculate Imout Indices

        float randnum;
        float this_px_out=0;
        float float_temp;
        randnum  = random[threadlinidx];
        int chan_idx=0;
        int chan=0;
        float dzdoutthis;
        float cur_lfilt;
        float cur_pfilt;
        float cur_in;
        size_t dzdout_idx = calcLinInd4(im_b_idx,filt_n_idx,im_h_idx,im_w_idx ,inp_b, filt_n, inp_h, inp_w);
        size_t dzdl_filt_idx;
        dzdoutthis = dzdout[dzdout_idx];
        for (chan =0 ; chan < inp_c ; chan++){
            dzdl_filt_idx = calcLinInd4(filt_n_idx , chan , filt_h_idx , filt_w_idx, filt_n, inp_c, filt_h, filt_w );
            cur_lfilt =  lfilt[dzdl_filt_idx];
            cur_pfilt = expf(cur_lfilt);
            if (cur_pfilt >= randnum){
                chan_idx = chan;
                break;
            }
            randnum = randnum - cur_pfilt;

        }
            size_t dzdin_idx = calcLinInd4(im_b_idx , chan_idx , inp_h_idx , inp_w_idx , inp_b, inp_c, inp_h, inp_w);
            cur_in = input[dzdin_idx];
            atomicAdd(&(dzdin[dzdin_idx]), (dzdoutthis));
            atomicAdd(&(dzdl_filt[dzdl_filt_idx]), dzdoutthis * (cur_in - 1 - cur_lfilt)); // p differentiable grad

    }



}

// End Kernels *************************

//Forward wrapper ----------------------
std::vector<at::Tensor> klconvs_cuda_forward(
at::Tensor input,
at::Tensor log_filt){

        //at::Tensor p_filt = (at::exp(log_filt));
       // p_filt = p_filt.cumsum(1);
        const auto batch_sz = input.size(0);
        const auto im_height = input.size(2);
        const auto im_width = input.size(3);
        const auto im_nchans = input.size(1);

        const auto filt_num = log_filt.size(0);
        const auto filt_height = log_filt.size(2);
        const auto filt_width = log_filt.size(3);
        //printf("(%d,%d,%d,%d)\n", p_filt.size(0),p_filt.size(1),p_filt.size(2),p_filt.size(3));
        //printf("filt_num:%d ",filt_num);
        auto out = at::zeros(input.type(),{batch_sz,filt_num,im_height,im_width}); //TODO: Remove except zero loop

        auto random = at::rand(input.type(),{filt_height,filt_width,batch_sz,filt_num,im_height,im_width});
        const int totalOutPx = im_height*im_width*batch_sz*filt_num;
        // Single Loop const auto totalThreads = totalOutPx*filt_height*filt_width;
        const  int totalThreads = totalOutPx;
        int j = 32;
        const  int threadsperblock =j*32;
        int blockNum = (totalThreads/threadsperblock);
        if (totalThreads%threadsperblock != 0 ){
            blockNum++;

        }
        const dim3 blocks(blockNum);
        //printf("blocks: %d, totaltherads/threadperbloc : %d", blocks,totalThreads/threadsperblock);

        AT_DISPATCH_FLOATING_TYPES(input.type(), "klconvs_forward_cuda", ([&] {
           klconvs_cuda_forward_kernel<scalar_t><<<blocks, threadsperblock>>>(
              input.data<scalar_t>(),
              log_filt.data<float>(),
              out.data<scalar_t>(),
              random.data<scalar_t>(),// rand . data please fix
              filt_width,
              filt_height,
              filt_num,
              im_width,
              im_height,
              im_nchans,
              batch_sz,
              totalOutPx,
              totalThreads

                );
          }));


        //out = out.sum(0);  /// ZEro Loop Version \TODO: rremove in case of diff kernel
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        return {out,random};

}
//----------------------------------------------
// Backward wrapper
std::vector<at::Tensor> klconvs_cuda_backward(at::Tensor dzdout,
at::Tensor input,
at::Tensor log_filt,
at::Tensor random
){

const auto batch_sz = input.size(0);
const auto im_height = input.size(2);
const auto im_width = input.size(3);
const auto im_nchans = input.size(1);

const auto filt_num = log_filt.size(0);
const auto filt_height = log_filt.size(2);
const auto filt_width = log_filt.size(3);

auto dzdinput = at::zeros_like(input);
auto dzdlfilt = at::zeros_like(log_filt);


// Single Loop const auto totalThreads = totalOutPx*filt_height*filt_width;
const  int totalThreads = im_height*im_width*batch_sz*filt_num*filt_height*filt_width;
int j = 32;
const  int threadsperblock =j*32;
int blockNum = (totalThreads/threadsperblock);
if (totalThreads%threadsperblock != 0 ){
    blockNum++;
}
const dim3 blocks(blockNum);
//printf("blocks: %d, totaltherads/threadperbloc : %d", blocks,totalThreads/threadsperblock);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "klconvs_backward_cuda", ([&] {
       klconvs_cuda_backward_kernel<scalar_t><<<blocks, threadsperblock>>>(  // TODO : CHANGE KLCONVS AND KLCONV BACK AND FORTH. NO FORGET.... NEVER FORGET, it is easy not to seee.
          input.data<scalar_t>(),
          log_filt.data<scalar_t>(),
          dzdout.data<scalar_t>(),
          random.data<scalar_t>(),// rand . data please fix
          dzdinput.data<float>(),
          dzdlfilt.data<float>(),
          filt_width,
          filt_height,
          filt_num,
          im_width,
          im_height,
          im_nchans,
          batch_sz,
          totalThreads
            );
      }));

        //dzdlfilt = at::div(dzdlfilt,im_height*im_width);
        //out = out.sum(0);  /// ZEro Loop Version \TODO: rremove in case of diff kernel
        gpuErrchk( hipPeekAtLastError() );
        //gpuErrchk( hipDeviceSynchronize() );



return {dzdinput, dzdlfilt};
}
std::vector<at::Tensor> klconvs_cuda_backward_rand(at::Tensor dzdout,
at::Tensor input,
at::Tensor log_filt
){

const auto batch_sz = input.size(0);
const auto im_height = input.size(2);
const auto im_width = input.size(3);
const auto im_nchans = input.size(1);

const auto filt_num = log_filt.size(0);
const auto filt_height = log_filt.size(2);
const auto filt_width = log_filt.size(3);

auto dzdinput = at::zeros_like(input);
auto dzdlfilt = at::zeros_like(log_filt);

auto random = at::rand(input.type(),{filt_height,filt_width,batch_sz,filt_num,im_height,im_width});
// Single Loop const auto totalThreads = totalOutPx*filt_height*filt_width;
const  int totalThreads = im_height*im_width*batch_sz*filt_num*filt_height*filt_width;
int j = 32;
const  int threadsperblock =j*32;
int blockNum = (totalThreads/threadsperblock);
if (totalThreads%threadsperblock != 0 ){
    blockNum++;
}
const dim3 blocks(blockNum);
//printf("blocks: %d, totaltherads/threadperbloc : %d", blocks,totalThreads/threadsperblock);

    AT_DISPATCH_FLOATING_TYPES(input.type(), "klconvs_backward_rand_cuda", ([&] {
       klconvs_cuda_backward_rand_kernel<scalar_t><<<blocks, threadsperblock>>>(  // TODO : CHANGE KLCONVS AND KLCONV BACK AND FORTH. NO FORGET.... NEVER FORGET, it is easy not to seee.
          input.data<scalar_t>(),
          log_filt.data<scalar_t>(),
          dzdout.data<scalar_t>(),
          random.data<scalar_t>(),// rand . data please fix
          dzdinput.data<float>(),
          dzdlfilt.data<float>(),
          filt_width,
          filt_height,
          filt_num,
          im_width,
          im_height,
          im_nchans,
          batch_sz,
          totalThreads
            );
      }));

        //dzdlfilt = at::div(dzdlfilt,im_height*im_width);
        //out = out.sum(0);  /// ZEro Loop Version \TODO: rremove in case of diff kernel
        gpuErrchk( hipPeekAtLastError() );
        //gpuErrchk( hipDeviceSynchronize() );



return {dzdinput, dzdlfilt};
}